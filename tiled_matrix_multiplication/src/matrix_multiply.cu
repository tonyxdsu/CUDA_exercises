#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../include/matrix_multiply.cuh"

__global__ void multiplyKernel(float* matrix1, float* matrix2, float* res, int heightRes, int widthRes, int width1height2) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float tile1[TILE_HEIGHT][TILE_WIDTH];
    __shared__ float tile2[TILE_WIDTH][TILE_HEIGHT];

    float sum = 0;

    int numTilesInMatrix1Width = width1height2 / TILE_WIDTH;
    if (width1height2 % TILE_WIDTH != 0) {
        numTilesInMatrix1Width++;
    }

    for (int k = 0; k < numTilesInMatrix1Width; k++) {
        if (row < heightRes && k * blockDim.x + threadIdx.x < width1height2) {
            tile1[threadIdx.y][threadIdx.x] = matrix1[row * width1height2 + k * blockDim.x + threadIdx.x];
        }
        else {
            tile1[threadIdx.y][threadIdx.x] = 0;
        }

        if (k * blockDim.y + threadIdx.y < width1height2 && col < widthRes) {
            tile2[threadIdx.y][threadIdx.x] = matrix2[(k * blockDim.y + threadIdx.y) * widthRes + col]; 
        }
        else {
            tile2[threadIdx.y][threadIdx.x] = 0; 
        }

        __syncthreads();
        
        for (int xTile1 = 0; xTile1 < TILE_WIDTH; xTile1++) {
            sum += tile1[threadIdx.y][xTile1] * tile2[xTile1][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < heightRes && col < widthRes) {
        res[row * widthRes + col] = sum;
    }
}

void matrixMultiply(float* matrix1_h, float* matrix2_h, float* matrixCalculatedRes_h, int height1, int width1, int height2, int width2) {
    hipError_t cudaStatus;

    float* matrix1_d = 0;
    float* matrix2_d = 0;
    float* matrixCalculatedRes_d = 0;

    cudaStatus = hipMalloc(&matrix1_d, height1 * width1 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrix1_d error");
    }

    cudaStatus = hipMalloc(&matrix2_d, height2 * width2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrix2_d error");
    }

    cudaStatus = hipMalloc(&matrixCalculatedRes_d, height1 * width2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrixCalculatedRes_d error");
    }

    cudaStatus = hipMemcpy(matrix1_d, matrix1_h, height1 * width1 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrix1_d error");
    }

    cudaStatus = hipMemcpy(matrix2_d, matrix2_h, height2 * width2 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrix2_h error");
    }

    int gridX = width2 / TILE_WIDTH;
    if (width2 % TILE_WIDTH != 0) {
        gridX += 1;
    }

    int gridY = height1 / TILE_HEIGHT;
    if (height1 % TILE_HEIGHT != 0) {
        gridY +=1;
    }

    dim3 dimGrid(gridX, gridY, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT, 1);

    // TODO occupancy calculator
    multiplyKernel<<<dimGrid, dimBlock>>>(matrix1_d, matrix2_d, matrixCalculatedRes_d, height1, width2, width1);
    
    cudaStatus = hipMemcpy(matrixCalculatedRes_h, matrixCalculatedRes_d, height1 * width2 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrixCalculatedRes_h error");
    }

    cudaStatus = hipFree(matrix1_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrix1_d error");
    }

    cudaStatus = hipFree(matrix2_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrix2_d error");
    }

    cudaStatus = hipFree(matrixCalculatedRes_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrixCalculatedRes_d error");
    }
}

