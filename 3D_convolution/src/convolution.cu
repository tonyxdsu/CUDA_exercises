#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../include/convolution.cuh"

__global__ void convolutionKernel(Tensor3D* input, Tensor3D* mask, Tensor3D* output) {
    int column = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int aisle = blockDim.z * blockIdx.z + threadIdx.z;

    int maskDimDivide2 = mask->xDim / 2;

    // TODO no way explicitly storing input->xyzDim in variable is faster right?
    // will be kept in register for every iteration right?

    if (column < input->xDim && row < input->yDim && aisle < input->zDim) {

        float sum = 0;

        for (int z = -maskDimDivide2; z <= maskDimDivide2; z++) {
            for (int y = -maskDimDivide2; y <= maskDimDivide2; y++) {
                for (int x = -maskDimDivide2; x <= maskDimDivide2; x++) {
                    if (x + column >= 0 && x + column < input->xDim &&
                        y + row >= 0 && y + row < input->yDim &&
                        z + aisle >= 0 && z + aisle < input->zDim) {
                        
                        sum += input->elements[(z + aisle) * input->xDim * input->yDim + (y + row) * input->xDim + (x + column)] *
                            mask->elements[(z + maskDimDivide2) * mask->xDim * mask->yDim + (y + maskDimDivide2) * mask->xDim + (x + maskDimDivide2)];
                    }
                }
            }
        }

        output->elements[aisle * input->xDim * input->yDim + row * input->xDim + column] = sum;
    }
}

Tensor3D* convolution(Tensor3D* input, Tensor3D* mask) {
    Tensor3D* output = new Tensor3D(input->xDim, input->yDim, input->zDim);

    unsigned int gridX = input->xDim / BLOCK_DIM;
    if (input->xDim % BLOCK_DIM != 0) {
        gridX++;
    }

    unsigned int gridY = input->yDim / BLOCK_DIM;
    if (input->yDim % BLOCK_DIM != 0) {
        gridY++;
    }

    unsigned int gridZ = input->zDim / BLOCK_DIM;
    if (input->zDim % BLOCK_DIM != 0) {
        gridZ++;
    }

    // TODO occupancy calculation
    dim3 dimGrid(gridX, gridY, gridZ);
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM, BLOCK_DIM);

    // TODO calculate available resources (ie registers per block) or kernel does not run with no error messages
    // TODO how to get error messages for kernel not running? nvprof?
    convolutionKernel<<<dimGrid, dimBlock>>>(input, mask, output);

    // TODO kernel calls are async so does this function return early without synchronize?
    hipDeviceSynchronize();

    return output;
}

