#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../include/convolution.cuh"

__global__ void convolutionKernel(float* matrix1, float* matrix2, float* res, int heightRes, int widthRes, int width1height2) {

}

void convolution(float* matrix1_h, float* matrix2_h, float* matrixCalculatedRes_h, int height1, int width1, int height2, int width2) {
    hipError_t cudaStatus;

    float* matrix1_d = 0;
    float* matrix2_d = 0;
    float* matrixCalculatedRes_d = 0;

    cudaStatus = hipMalloc(&matrix1_d, height1 * width1 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrix1_d error");
    }

    cudaStatus = hipMalloc(&matrix2_d, height2 * width2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrix2_d error");
    }

    cudaStatus = hipMalloc(&matrixCalculatedRes_d, height1 * width2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc matrixCalculatedRes_d error");
    }

    cudaStatus = hipMemcpy(matrix1_d, matrix1_h, height1 * width1 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrix1_d error");
    }

    cudaStatus = hipMemcpy(matrix2_d, matrix2_h, height2 * width2 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrix2_h error");
    }

    int gridX = width2 / BLOCK_DIM;
    if (width2 % BLOCK_DIM != 0) {
        gridX += 1;
    }

    int gridY = height1 / BLOCK_DIM;
    if (height1 % BLOCK_DIM != 0) {
        gridY +=1;
    }

    dim3 dimGrid(gridX, gridY, 1);
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM, 1);

    // TODO occupancy calculator
    convolutionKernel<<<dimGrid, dimBlock>>>(matrix1_d, matrix2_d, matrixCalculatedRes_d, height1, width2, width1);
    
    cudaStatus = hipMemcpy(matrixCalculatedRes_h, matrixCalculatedRes_d, height1 * width2 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy matrixCalculatedRes_h error");
    }

    cudaStatus = hipFree(matrix1_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrix1_d error");
    }

    cudaStatus = hipFree(matrix2_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrix2_d error");
    }

    cudaStatus = hipFree(matrixCalculatedRes_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree matrixCalculatedRes_d error");
    }
}

