#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../include/vector_addition.cuh"

__global__ void additionKernel(float* vector1, float* vector2, float* res) {
    int i = threadIdx.x;
    res[i] = vector1[i] + vector2[i];
}

void vectorAddition(float* vector1_h, float* vector2_h, float* vectorCalculatedRes_h, int height, int width) {
    hipError_t cudaStatus;

    float* vector1_d = 0;
    float* vector2_d = 0;
    float* vectorCalculatedRes_d = 0;

    cudaStatus = hipMalloc(&vector1_d, height * width * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc vector1_d error");
    }

    cudaStatus = hipMalloc(&vector2_d, height * width * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc vector2_d error");
    }

    cudaStatus = hipMalloc(&vectorCalculatedRes_d, height * width * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc vectorCalculatedRes_d error");
    }

    cudaStatus = hipMemcpy(vector1_d, vector1_h, height * width * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy vector1_d error");
    }

    cudaStatus = hipMemcpy(vector2_d, vector2_h, height * width * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy vector2_h error");
    }

    // TODO occupancy calculator
    additionKernel<<<1, height * width>>>(vector1_d, vector2_d, vectorCalculatedRes_d);
    
    cudaStatus = hipMemcpy(vectorCalculatedRes_h, vectorCalculatedRes_d, height * width * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy vectorCalculatedRes_h error");
    }

    cudaStatus = hipFree(vector1_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree vector1_d error");
    }

    cudaStatus = hipFree(vector2_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree vector2_d error");
    }

    cudaStatus = hipFree(vectorCalculatedRes_d);
    if (cudaStatus != hipSuccess) {
        printf("hipFree vectorCalculatedRes_d error");
    }
}

